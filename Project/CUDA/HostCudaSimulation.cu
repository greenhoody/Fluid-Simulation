#include "hip/hip_runtime.h"
#include "HostCudaSimulation.h"
#include "DeviceCudaSimulation.cuh"

#include <hip/hip_runtime_api.h>

HostCudaSimulation::HostCudaSimulation(int size, float diffiusion, float viscosity, float dt):Simulation(size,diffiusion,viscosity,dt)
{	
	// w cuda nie ma odpowiednika calloc
	hipMalloc(&u, sizeof(float) * n);
	hipMalloc(&v, sizeof(float) * n);
	hipMalloc(&u_prev, sizeof(float) * n);
	hipMalloc(&v_prev, sizeof(float) * n);
	hipMalloc(&dens, sizeof(float) * n);
	hipMalloc(&dens_prev, sizeof(float) * n);

	hipMemset(u, 0, sizeof(float) * n);
	hipMemset(v, 0, sizeof(float) * n);
	hipMemset(u_prev, 0, sizeof(float) * n);
	hipMemset(v_prev, 0, sizeof(float) * n);
	hipMemset(dens, 0, sizeof(float) * n);
	hipMemset(dens_prev, 0, sizeof(float) * n);
}

HostCudaSimulation::~HostCudaSimulation()
{
	hipFree(u);
	hipFree(v);
	hipFree(u_prev);
	hipFree(v_prev);
	hipFree(dens);
	hipFree(dens_prev);
}

void HostCudaSimulation::NextFrame(float* copy_array) 
{


	void* kernelArgs[] = {&size, &dens, &dens_prev, &u, &v, &u_prev, &v_prev, &visc, &diff, &dt };

	int numBlocksPerSm = 0;
	// Number of threads my_kernel will be launched with
	int numThreads = 32;
	hipDeviceProp_t deviceProp;
	// device zero ponieważ to zzazwyczaj najwydajniesza karta, w moim przypadku jest tylko jedna karta
	hipGetDeviceProperties(&deviceProp, 0);
	hipError_t e = hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, cuda_NextFrame, numThreads, 0);
	dim3 blockDim(numThreads, 1, 1);
	dim3 gridDim(deviceProp.multiProcessorCount * numBlocksPerSm, 1, 1);

	hipLaunchCooperativeKernel((void*)cuda_NextFrame, gridDim, blockDim, kernelArgs);

	hipDeviceSynchronize();
	hipMemcpy(copy_array, dens, sizeof(float) * n, hipMemcpyDeviceToHost);
}

void HostCudaSimulation::AddDensity(int x1, int x2, int y1, int y2, float density)
{
	if (x1 > x2)
	{
		int tmp = x1;
		x1 = x2;
		x2 = tmp;
	}
	if (y1 > y2)
	{
		int tmp = y1;
		y1 = y2;
		y2 = tmp;
	}

	x1 = x1 < 0 ? 0 : x1;
	y1 = y1 < 0 ? 0 : y1;
	x2 = x2 > size ? size : x2;
	y2 = y2 > size ? size : y2;

	int delta_x = x2 - x1;
	int delta_y = y2 - y1;
	int cells = delta_x * delta_y;

	int blocks = ceilf((float)cells / 1024);
	dim3 gridDim = dim3(blocks, 1, 1);
	dim3 blockDim = dim3(1024, 1, 1);


	void* kernelArgs[] = { &size, &dens, &x1, &x2, &y1, &y2, &density };

	hipLaunchCooperativeKernel((void*)addDensity, gridDim, blockDim, kernelArgs);
	hipDeviceSynchronize();
}

void HostCudaSimulation::AddVelocity(int x, int y, int r, float v_velocity, float h_velocity)
{
	int cells = 4 * r * r;
	int blocks = ceilf((float)cells / 1024);
	dim3 blockDim;
	dim3 gridDim = dim3(blocks, 1, 1);
	if (blocks == 1) {
		blockDim = dim3(cells, 1, 1);
	}
	else
	{
		blockDim = dim3(1024, 1, 1);
	}

	void* kernelArgs[] = { &size,&u,&v, &x, &y, &r, &h_velocity, &v_velocity };

	hipLaunchCooperativeKernel((void*)addVelocity, gridDim, blockDim, kernelArgs);
	hipDeviceSynchronize();
}