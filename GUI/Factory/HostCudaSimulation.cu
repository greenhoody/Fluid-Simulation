#include "hip/hip_runtime.h"
#include"pch.h"
#include "HostCudaSimulation.cuh"
#include "DeviceCudaSimulation.cuh"

#include <hip/hip_runtime_api.h>

HostCudaSimulation::HostCudaSimulation(int size, float diffiusion, float viscosity, float dt):Simulation(size,diffiusion,viscosity,dt)
{	
	// w cuda nie ma odpowiednika calloc
	
	hipMalloc(&u, sizeof(float) * n);
	hipMalloc(&v, sizeof(float) * n);
	hipMalloc(&u_prev, sizeof(float) * n);
	hipMalloc(&v_prev, sizeof(float) * n);
	hipMalloc(&dens, sizeof(float) * n);
	hipMalloc(&dens_prev, sizeof(float) * n);

	hipMalloc(&u_const, sizeof(float) * n);
	hipMalloc(&v_const, sizeof(float) * n);
	hipMalloc(&dens_const, sizeof(float) * n);

	hipMemset(u, 0, sizeof(float) * n);
	hipMemset(v, 0, sizeof(float) * n);
	hipMemset(u_prev, 0, sizeof(float) * n);
	hipMemset(v_prev, 0, sizeof(float) * n);
	hipMemset(dens, 0, sizeof(float) * n);
	hipMemset(dens_prev, 0, sizeof(float) * n);

	hipMemset(u_const, 0, sizeof(float) * n);
	hipMemset(v_const, 0, sizeof(float) * n);
	hipMemset(dens_const, 0, sizeof(float) * n);
}
HostCudaSimulation::~HostCudaSimulation()
{
	hipFree(u);
	hipFree(v);
	hipFree(u_prev);
	hipFree(v_prev);
	hipFree(dens);
	hipFree(dens_prev);

	hipFree(u_const);
	hipFree(v_const);
	hipFree(dens_const);
}

void HostCudaSimulation::NextFrame(std::shared_ptr<float[]> copy_array) 
{

	//int blocks = ceilf((float)(size * size) / 1024);
	//dim3 gridDim = dim3(blocks,1,1);
	//dim3 blockDim = dim3(1024, 1, 1);
	void* kernelArgs[] = {&size, &dens, &dens_prev, &u, &v, &u_prev, &v_prev, &visc, &diff, &dt };



	int numBlocksPerSm = 0;
	// Number of threads my_kernel will be launched with
	int numThreads = 128;
	hipDeviceProp_t deviceProp;
	// device zero poniewa� jest tylko jedna karta
	hipGetDeviceProperties(&deviceProp, 0);
	hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, cuda_NextFrame, numThreads, 0);
	dim3 blockDim(numThreads, 1, 1);
	dim3 gridDim(deviceProp.multiProcessorCount * numBlocksPerSm, 1, 1);

	hipLaunchCooperativeKernel((void*)cuda_NextFrame, gridDim, blockDim, kernelArgs);

	//cuda_NextFrame<<< gridDim, blockDim>>>(size, dens, dens_prev, u, v, u_prev, v_prev,visc,diff,dt);

	hipDeviceSynchronize();
	hipMemcpy(copy_array.get(), dens, sizeof(float) * n, hipMemcpyDeviceToHost);
}

void HostCudaSimulation::AddDensity(int x1, int x2, int y1, int y2, float density)
{
	if (x1 > x2)
	{
		int tmp = x1;
		x1 = x2;
		x2 = tmp;
	}
	if (y1 > y2)
	{
		int tmp = y1;
		y1 = y2;
		y2 = tmp;
	}

	x1 = x1 < 0 ? 0 : x1;
	y1 = y1 < 0 ? 0 : y1;
	x2 = x2 > size ? size : x2;
	y2 = y2 > size ? size : y2;

	int delta_x = x2 - x1;
	int delta_y = y2 - y1;
	int cells = delta_x * delta_y;

	int blocks = ceilf((float)cells / 1024);
	dim3 gridDim = dim3(blocks, 1, 1);
	dim3 blockDim = dim3(1024, 1, 1);


	void* kernelArgs[] = { &size, &dens, &x1, &x2, &y1, &y2, &density };

	hipLaunchCooperativeKernel((void*)addDensity, gridDim, blockDim, kernelArgs);

	//addDensity<<<gridDim, blockDim>>>(size,dens,x1,x2,y1,y2,density);
	hipDeviceSynchronize();
}

void HostCudaSimulation::AddVelocity(int x, int y, int r, float v_velocity, float h_velocity)
{
	int cells = 4 * r * r;
	int blocks = ceilf((float)cells / 1024);
	dim3 blockDim;
	dim3 gridDim = dim3(blocks, 1, 1);
	if (blocks == 1) {
		blockDim = dim3(cells, 1, 1);
	}
	else
	{
		blockDim = dim3(1024, 1, 1);
	}

	

	void* kernelArgs[] = { &size,&u,&v, &x, &y, &r, &h_velocity, &v_velocity };

	hipLaunchCooperativeKernel((void*)addVelocity, gridDim, blockDim, kernelArgs);
	//addVelocity<<<gridDim, blockDim>>>(size,u,v,x,y,r,h_velocity,v_velocity);
	hipDeviceSynchronize();
}

void HostCudaSimulation::AddConstantDensity(int x, int y, float density) {}
void HostCudaSimulation::DeleteConstantDensity(int x, int y){}
void HostCudaSimulation::AddConstantVelocity(int x, int y, float v_velocity, float h_velocity){}
void HostCudaSimulation::DeleteConstantVelocity(int x, int y){}