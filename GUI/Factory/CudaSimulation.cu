#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include "CudaSimulation.h"

#define IX(i,j) ((i)+(size+2)*(j))

CudaSimulation::CudaSimulation(int size, float diffiusion, float viscosity, float dt) : Simulation(size, diffiusion, viscosity, dt) {
	n = (size + 2) * (size + 2);
	
	block_no = (n / 1024);

	// w cuda nie ma odpowiednika calloc
	
	hipMalloc(&d_u, sizeof(float) * n);
	hipMalloc(&d_v, sizeof(float) * n);
	hipMalloc(&d_u_prev, sizeof(float) * n);
	hipMalloc(&d_v_prev, sizeof(float) * n);
	hipMalloc(&d_dens, sizeof(float) * n);
	hipMalloc(&d_dens_prev, sizeof(float) * n);

	hipMalloc(&d_u_const, sizeof(float) * n);
	hipMalloc(&d_v_const, sizeof(float) * n);
	hipMalloc(&d_dens_const, sizeof(float) * n);

	hipMemset(d_u, 0, sizeof(float) * n);
	hipMemset(d_v, 0, sizeof(float) * n);
	hipMemset(d_u_prev, 0, sizeof(float) * n);
	hipMemset(d_v_prev, 0, sizeof(float) * n);
	hipMemset(d_dens, 0, sizeof(float) * n);
	hipMemset(d_dens_prev, 0, sizeof(float) * n);

	hipMemset(d_u_const, 0, sizeof(float) * n);
	hipMemset(d_v_const, 0, sizeof(float) * n);
	hipMemset(d_dens_const, 0, sizeof(float) * n);
	
}

CudaSimulation::~CudaSimulation() {
	hipFree(d_u);
	hipFree(d_v);
	hipFree(d_u_prev);
	hipFree(d_v_prev);
	hipFree(d_dens);
	hipFree(d_dens_prev);

	hipFree(d_u_const);
	hipFree(d_v_const);
	hipFree(d_dens_const);
}

void CudaSimulation::NextFrame(std::shared_ptr<float[]> copy_array) {
	
	cuda_NextFrame();
	hipMemcpy(copy_array.get(), d_dens, sizeof(float) * (size + 2) * (size + 2), hipMemcpyDeviceToHost);
}

void CudaSimulation::AddDensity(int x, int y, float density) {
	int index = IX(x + 1, y + 1);
	dens[index] += density;
	if (dens[index] > 1) {
		dens[index] = 1;
	}
}

void CudaSimulation::AddVelocity(int x, int y, float h_velocity, float v_velocity) {
	int index = IX(x + 1, y + 1);
	//v z indeksem ujemnym wpisuje dane do u_prev/ mazanie po pamięci
	v[index] += v_velocity;
	u[index] += h_velocity;
}

void CudaSimulation::AddConstantDensity(int x, int y, float density) {}
void CudaSimulation::DeleteConstantDensity(int x, int y) {}
void CudaSimulation::AddConstantVelocity(int x, int y, float v_velocity, float h_velocity) {}
void CudaSimulation::DeleteConstantVelocity(int x, int y) {}
//=====================================================================================================================

void CudaSimulation::add_source(int N, float* x, float* s, float dt)
{
	int i, size = (N + 2) * (N + 2);
	for (i = 0; i < size; i++) x[i] += dt * s[i];
}

__device__ void CudaSimulation::set_bnd(int N, int b, float* x)
{
	int i;
	for (i = 1; i <= N; i++) {
		x[IX(0, i)] = b == 1 ? -x[IX(1, i)] : x[IX(1, i)];
		x[IX(N + 1, i)] = b == 1 ? -x[IX(N, i)] : x[IX(N, i)];
		x[IX(i, 0)] = b == 2 ? -x[IX(i, 1)] : x[IX(i, 1)];
		x[IX(i, N + 1)] = b == 2 ? -x[IX(i, N)] : x[IX(i, N)];
	}
	x[IX(0, 0)] = 0.5f * (x[IX(1, 0)] + x[IX(0, 1)]);
	x[IX(0, N + 1)] = 0.5f * (x[IX(1, N + 1)] + x[IX(0, N)]);
	x[IX(N + 1, 0)] = 0.5f * (x[IX(N, 0)] + x[IX(N + 1, 1)]);
	x[IX(N + 1, N + 1)] = 0.5f * (x[IX(N, N + 1)] + x[IX(N + 1, N)]);
}

__device__ void CudaSimulation::diffuse(int N, int b, float* x, float* x0, float diff, float dt)
{
	int i, j, k;
	float a = dt * diff * N * N;
	for (k = 0; k < 20; k++) {
		for (i = 1; i <= N; i++) {
			for (j = 1; j <= N; j++) {
				x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] +
					x[IX(i, j - 1)] + x[IX(i, j + 1)])) / (1 + (4 * a));
			}
		}
		set_bnd(N, b, x);
	}
}

__device__ void CudaSimulation::advect(int N, int b, float* d, float* d0, float* u, float* v, float dt)
{
	int i0, j0, i1, j1;
	float x, y, s0, t0, s1, t1, dt0;
	dt0 = dt * (float)N;
	for (int i = 1; i <= N; i++) {
		for (int j = 1; j <= N; j++) {

			x = (float)i - dt0 * u[IX(i, j)];
			if (x < 0.5) x = 0.5f;
			if (x > N + 0.5) x = N + 0.5f;
			i0 = (int)x;
			i1 = i0 + 1;

			//proporcje ile gę┌stości wylądowało z których komórek
			s1 = x - (float)i0;
			s0 = 1.0f - s1;

			y = (float)j - dt0 * v[IX(i, j)];
			if (y < 0.5) y = 0.5f;
			if (y > N + 0.5) y = N + 0.5f;
			j0 = (int)y;
			j1 = j0 + 1;

			//proporcje ile gęstości wylądowało z których komórek
			t1 = y - (float)j0;
			t0 = 1.0f - t1;

			d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
				s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);

		}
	}
	set_bnd(N, b, d);
}

__device__ void CudaSimulation::project(int N, float* u, float* v, float* p, float* div)
{
	int i, j, k;
	float h;
	h = 1.0f / N;
	for (i = 1; i <= N; i++) {
		for (j = 1; j <= N; j++) {
			div[IX(i, j)] = -0.5f * h * (u[IX(i + 1, j)] - u[IX(i - 1, j)] +
				v[IX(i, j + 1)] - v[IX(i, j - 1)]);
			p[IX(i, j)] = 0.0f;
		}
	}
	set_bnd(N, 0, div);
	set_bnd(N, 0, p);


	for (k = 0; k < 20; k++) {
		for (i = 1; i <= N; i++) {
			for (j = 1; j <= N; j++) {
				p[IX(i, j)] = (div[IX(i, j)] + p[IX(i - 1, j)] + p[IX(i + 1, j)] +
					p[IX(i, j - 1)] + p[IX(i, j + 1)]) / 4;
			}
		}
		set_bnd(N, 0, p);
	}
	for (i = 1; i <= N; i++) {
		for (j = 1; j <= N; j++) {
			u[IX(i, j)] -= 0.5f * (p[IX(i + 1, j)] - p[IX(i - 1, j)]) / h;
			v[IX(i, j)] -= 0.5f * (p[IX(i, j + 1)] - p[IX(i, j - 1)]) / h;
		}
	}
	set_bnd(N, 1, u);
	set_bnd(N, 2, v);
}

__device__ void CudaSimulation::vel_step(int N, float* u, float* v, float* u0, float* v0, float visc, float dt)
{
	//add_source(N, u, u0, dt); add_source(N, v, v0, dt);
	diffuse(N, 1, u0, u, visc, dt);
	diffuse(N, 2, v0, v, visc, dt);
	project(N, u0, v0, u, v);
	advect(N, 1, u, u0, u0, v0, dt);
	advect(N, 2, v, v0, u0, v0, dt);
	project(N, u, v, u0, v0);
}

__device__ void CudaSimulation::dens_step(int N, float* x, float* x0, float* u, float* v, float diff, float dt)
{
	//add_source(N, x, x0, dt);
	//print(x, "poczatek:");
	diffuse(N, 0, x0, x, diff, dt);
	//print(x, "po dyfuzji:");
	//adwekcja zmienia ilość2 cieczy
	advect(N, 0, x, x0, u, v, dt);
	//print(x, "po advekcji:");
}

__global__ void CudaSimulation::cuda_NextFrame() {
	vel_step(size, d_u, d_v, d_u_prev, d_v_prev, visc, dt);
	dens_step(size, d_dens, d_dens_prev, d_u, d_v, diff, dt);
}