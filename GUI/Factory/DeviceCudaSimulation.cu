#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include "DeviceCudaSimulation.cuh"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime_api.h> 
#include <hip/hip_runtime.h>


#define IX(i,j) ((i)+(N+2)*(j))

//void CudaSimulation::add_source(int N, float* x, float* s, float dt)
//{
//	int i, size = (N + 2) * (N + 2);
//	for (i = 0; i < size; i++) x[i] += dt * s[i];
//}


// wątek zero rogi, reszta krawędzie, wiem że nie optymalne, ale może późjniej nad tym pomyśle i wywali się przy zamałej ilości wątków, na początku synchronizacja grid. Ta funkcja jest wywołyuwana wiele razy, więc wiele synchronizacji więc trzeba sprawdzić póxniej czy potrzebne
__device__ void set_bnd(int N, int b, float* x)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	cooperative_groups::grid_group g = cooperative_groups::this_grid();
	g.sync();

	if (index > 0 && index < N + 2) {
		x[IX(0, index)] = b == 1 ? -x[IX(1, index)] : x[IX(1, index)];
		x[IX(N + 1, index)] = b == 1 ? -x[IX(N, index)] : x[IX(N, index)];
		x[IX(index, 0)] = b == 2 ? -x[IX(index, 1)] : x[IX(index, 1)];
		x[IX(index, N + 1)] = b == 2 ? -x[IX(index, N)] : x[IX(index, N)];
	}
	
	//for (int i = 1; i <= N; i++) {
	//	x[IX(0, i)] = b == 1 ? -x[IX(1, i)] : x[IX(1, i)];
	//	x[IX(N + 1, i)] = b == 1 ? -x[IX(N, i)] : x[IX(N, i)];
	//	x[IX(i, 0)] = b == 2 ? -x[IX(i, 1)] : x[IX(i, 1)];
	//	x[IX(i, N + 1)] = b == 2 ? -x[IX(i, N)] : x[IX(i, N)];
	//}

	if (index == 0)
	{
		x[IX(0, 0)] = 0.5f * (x[IX(1, 0)] + x[IX(0, 1)]);
		x[IX(0, N + 1)] = 0.5f * (x[IX(1, N + 1)] + x[IX(0, N)]);
		x[IX(N + 1, 0)] = 0.5f * (x[IX(N, 0)] + x[IX(N + 1, 1)]);
		x[IX(N + 1, N + 1)] = 0.5f * (x[IX(N, N + 1)] + x[IX(N + 1, N)]);
	}

}

__device__ void diffuse(int N, int b, float* x, float* x0, float diff, float dt)
{
	int n = N * N;
	float a = dt * diff * n;
	
	
	int index = 0;
	int cores = gridDim.x * blockDim.x;

	//kompiluje się czyli intellisense nie ogarnia
	cooperative_groups::grid_group g = cooperative_groups::this_grid();

	for (int k = 0; k < 20; k++) {
		
		// wykonanie przypadających komurek
		index = blockIdx.x * blockDim.x + threadIdx.x;
		while (index < n) {
			// +1 po od 1 zaczynały się pętle, a nie od zera
			int j = (index / N) + 1;
			int i = (index % N) + 1;

			//for (int i = 1; i <= N; i++) {
			//	for (int j = 1; j <= N; j++) {
			x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] +
				x[IX(i, j - 1)] + x[IX(i, j + 1)])) / (1 + (4 * a));
			index += cores;
		}
		//	}
		//}
		g.sync();
		set_bnd(N, b, x);
	}
}

__device__ void advect(int N, int b, float* d, float* d0, float* u, float* v, float dt)
{
	cooperative_groups::grid_group g = cooperative_groups::this_grid();

	int n = N * N;
	int cores = gridDim.x * blockDim.x;
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	while(index < n)
	{
		// +1 po od 1 zaczynały się pętle, a nie od zera
		int j = (index / N) + 1;
		int i = (index % N) + 1;

		int i0, j0, i1, j1;
		float x, y, s0, t0, s1, t1, dt0;
		dt0 = dt * (float)N;
		//for (int i = 1; i <= N; i++) {
		//	for (int j = 1; j <= N; j++) {

		x = (float)i - dt0 * u[IX(i, j)];
		if (x < 0.5) x = 0.5f;
		if (x > N + 0.5) x = N + 0.5f;
		i0 = (int)x;
		i1 = i0 + 1;

		//proporcje ile gę┌stości wylądowało z których komórek
		s1 = x - (float)i0;
		s0 = 1.0f - s1;

		y = (float)j - dt0 * v[IX(i, j)];
		if (y < 0.5) y = 0.5f;
		if (y > N + 0.5) y = N + 0.5f;
		j0 = (int)y;
		j1 = j0 + 1;

		//proporcje ile gęstości wylądowało z których komórek
		t1 = y - (float)j0;
		t0 = 1.0f - t1;

		d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
			s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);

		//	}
		//}

		index += cores;
	}

	set_bnd(N, b, d);
}

__device__ void project(int N, float* u, float* v, float* p, float* div)
{

	cooperative_groups::grid_group g = cooperative_groups::this_grid();
	int n = N * N;
	int cores = blockDim.x * gridDim.x;

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	// +1 po od 1 zaczynały się pętle, a nie od zera


	float h = 1.0f / N;
	//for (i = 1; i <= N; i++) {
	//	for (j = 1; j <= N; j++) {
	while (index < n) {
		int j = (index / N) + 1;
		int i = (index % N) + 1;

		div[IX(i, j)] = -0.5f * h * (u[IX(i + 1, j)] - u[IX(i - 1, j)] +
			v[IX(i, j + 1)] - v[IX(i, j - 1)]);
		p[IX(i, j)] = 0.0f;

		index += cores;
	 }


	//	}
	//}

	g.sync();
	set_bnd(N, 0, div);
	set_bnd(N, 0, p);

	index = blockIdx.x * blockDim.x + threadIdx.x;
	for (int k = 0; k < 20; k++) {
		while (index < n)
		{
			int j = (index / N) + 1;
			int i = (index % N) + 1;

			p[IX(i, j)] = (div[IX(i, j)] + p[IX(i - 1, j)] + p[IX(i + 1, j)] +
				p[IX(i, j - 1)] + p[IX(i, j + 1)]) / 4;
			index += cores;

		}
		g.sync();
		set_bnd(N, 0, p);
	}

	index = blockIdx.x * blockDim.x + threadIdx.x;
	while (index < n)
	{
		int j = (index / N) + 1;
		int i = (index % N) + 1;

		u[IX(i, j)] -= 0.5f * (p[IX(i + 1, j)] - p[IX(i - 1, j)]) / h;
		v[IX(i, j)] -= 0.5f * (p[IX(i, j + 1)] - p[IX(i, j - 1)]) / h;
		index += cores;
	}

	g.sync();
	set_bnd(N, 1, u);
	set_bnd(N, 2, v);
}

__global__ void cuda_NextFrame(int N, float* d_dens, float* d_dens_prev, float* d_u, float* d_v, float* d_u_prev, float* d_v_prev, float visc, float diff , float dt) {
	//vel_step(size, d_u, d_v, d_u_prev, d_v_prev, visc, dt);

	diffuse(N, 1, d_u_prev, d_u, visc, dt);
	diffuse(N, 2, d_v_prev, d_v, visc, dt);
	project(N, d_u_prev, d_v_prev, d_u, d_v);
	advect(N, 1, d_u, d_u_prev, d_u_prev, d_v_prev, dt);
	advect(N, 2, d_v, d_v_prev, d_u_prev, d_v_prev, dt);
	project(N, d_u, d_v, d_u_prev, d_v_prev);
	
	
	//dens_step(size, d_dens, d_dens_prev, d_u, d_v, diff, dt);

	diffuse(N, 0, d_dens_prev, d_dens, diff, dt);
	advect(N, 0, d_dens, d_dens_prev, d_u, d_v, dt);
}

__global__ void addDensity(int N, float* d_dens, int x1, int x2, int y1, int y2, float dens)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int cores = blockDim.x * gridDim.x;
	int deltay = y2 - y1;
	int deltax = x2 - x1;
	int n = deltay * deltax;


	while (index < n) 
	{
		int j = (index / deltax);
		int i = (index % deltax);

		d_dens[IX(x1 + i, y1 + j)] += dens;

		if (d_dens[IX(x1 + i, y1 + j)] > 1)
		{
			d_dens[IX(x1 + i, y1 + j)] = 1;
		}
		index += cores;
	}
}

__global__ void addVelocity(int N, float* d_u, float* d_v, int x, int y, int r, float u_velocity, float v_velocity)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int cores = blockDim.x * gridDim.x;
	int j = (index / (r*2));
	int i = (index % (r*2));
	int r_square = r * r;
	int n = r_square * 4;
	x -= r;
	y -= r;

	while (index < n) {

		if (N > x + i && x + i > 0 && N > y + j && y + j > 0 && i * i + j * j <= r_square)
		{
			d_u[IX(x + i, y + j)] += u_velocity;
			d_v[IX(x + i, y + j)] += v_velocity;
		}

		index += cores;
	}


}
//__device__ void vel_step(int N, float* u, float* v, float* u0, float* v0, float visc, float dt)
//{
//	//add_source(N, u, u0, dt); add_source(N, v, v0, dt);
//	diffuse(N, 1, u0, u, visc, dt);
//	diffuse(N, 2, v0, v, visc, dt);
//	project(N, u0, v0, u, v);
//	advect(N, 1, u, u0, u0, v0, dt);
//	advect(N, 2, v, v0, u0, v0, dt);
//	project(N, u, v, u0, v0);
//}

//__device__ void dens_step(int N, float* x, float* x0, float* u, float* v, float diff, float dt)
//{
//	//add_source(N, x, x0, dt);
//	//print(x, "poczatek:");
//	diffuse(N, 0, x0, x, diff, dt);
//	//print(x, "po dyfuzji:");
//	//adwekcja zmienia ilość2 cieczy
//	advect(N, 0, x, x0, u, v, dt);
//	//print(x, "po advekcji:");
//}