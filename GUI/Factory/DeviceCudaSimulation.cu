#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include "DeviceCudaSimulation.cuh"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime_api.h> 
#include <hip/hip_runtime.h>


#define IX(i,j) ((i)+(N+2)*(j))

//CudaSimulation::CudaSimulation(int size, float diffiusion, float viscosity, float dt) : Simulation(size, diffiusion, viscosity, dt) {
//	n = (size + 2) * (size + 2);
//	
//	block_no = (n / 1024);
//
//	// w cuda nie ma odpowiednika calloc
//	
//	hipMalloc(&d_u, sizeof(float) * n);
//	hipMalloc(&d_v, sizeof(float) * n);
//	hipMalloc(&d_u_prev, sizeof(float) * n);
//	hipMalloc(&d_v_prev, sizeof(float) * n);
//	hipMalloc(&d_dens, sizeof(float) * n);
//	hipMalloc(&d_dens_prev, sizeof(float) * n);
//
//	hipMalloc(&d_u_const, sizeof(float) * n);
//	hipMalloc(&d_v_const, sizeof(float) * n);
//	hipMalloc(&d_dens_const, sizeof(float) * n);
//
//	hipMemset(d_u, 0, sizeof(float) * n);
//	hipMemset(d_v, 0, sizeof(float) * n);
//	hipMemset(d_u_prev, 0, sizeof(float) * n);
//	hipMemset(d_v_prev, 0, sizeof(float) * n);
//	hipMemset(d_dens, 0, sizeof(float) * n);
//	hipMemset(d_dens_prev, 0, sizeof(float) * n);
//
//	hipMemset(d_u_const, 0, sizeof(float) * n);
//	hipMemset(d_v_const, 0, sizeof(float) * n);
//	hipMemset(d_dens_const, 0, sizeof(float) * n);
//	
//}
//
//CudaSimulation::~CudaSimulation() {
//	hipFree(d_u);
//	hipFree(d_v);
//	hipFree(d_u_prev);
//	hipFree(d_v_prev);
//	hipFree(d_dens);
//	hipFree(d_dens_prev);
//
//	hipFree(d_u_const);
//	hipFree(d_v_const);
//	hipFree(d_dens_const);
//}

//void CudaSimulation::NextFrame(std::shared_ptr<float[]> copy_array) {
//	
//	int blocks = ceilf((float)n / 1024);
//	dim3 gridDim = dim3(blocks,1,1);
//	dim3 blockDim = dim3(1024, 1, 1);
//	void* kernelArgs = {};
//
//	hipLaunchCooperativeKernel(  (void*) CudaSimulation::cuda_NextFrame, gridDim, blockDim, kernelArgs);
//
//	cuda_NextFrame<<<blocks, 1024>>>();
//	hipMemcpy(copy_array.get(), d_dens, sizeof(float) * (size + 2) * (size + 2), hipMemcpyDeviceToHost);
//}


//=====================================================================================================================

//void CudaSimulation::add_source(int N, float* x, float* s, float dt)
//{
//	int i, size = (N + 2) * (N + 2);
//	for (i = 0; i < size; i++) x[i] += dt * s[i];
//}


// wątek zero rogi, reszta krawędzie, wiem że nie optymalne, ale może późjniej nad tym pomyśle i wywali się przy zamałej ilości wątków, na początku synchronizacja grid. Ta funkcja jest wywołyuwana wiele razy, więc wiele synchronizacji więc trzeba sprawdzić póxniej czy potrzebne
__device__ void set_bnd(int N, int b, float* x)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	cooperative_groups::grid_group g = cooperative_groups::this_grid();
	g.sync();

	if (index > 0 && index < N + 2) {
		x[IX(0, index)] = b == 1 ? -x[IX(1, index)] : x[IX(1, index)];
		x[IX(N + 1, index)] = b == 1 ? -x[IX(N, index)] : x[IX(N, index)];
		x[IX(index, 0)] = b == 2 ? -x[IX(index, 1)] : x[IX(index, 1)];
		x[IX(index, N + 1)] = b == 2 ? -x[IX(index, N)] : x[IX(index, N)];
	}
	
	//for (int i = 1; i <= N; i++) {
	//	x[IX(0, i)] = b == 1 ? -x[IX(1, i)] : x[IX(1, i)];
	//	x[IX(N + 1, i)] = b == 1 ? -x[IX(N, i)] : x[IX(N, i)];
	//	x[IX(i, 0)] = b == 2 ? -x[IX(i, 1)] : x[IX(i, 1)];
	//	x[IX(i, N + 1)] = b == 2 ? -x[IX(i, N)] : x[IX(i, N)];
	//}

	if (index == 0)
	{
		x[IX(0, 0)] = 0.5f * (x[IX(1, 0)] + x[IX(0, 1)]);
		x[IX(0, N + 1)] = 0.5f * (x[IX(1, N + 1)] + x[IX(0, N)]);
		x[IX(N + 1, 0)] = 0.5f * (x[IX(N, 0)] + x[IX(N + 1, 1)]);
		x[IX(N + 1, N + 1)] = 0.5f * (x[IX(N, N + 1)] + x[IX(N + 1, N)]);
	}

}

__device__ void diffuse(int N, int b, float* x, float* x0, float diff, float dt)
{
	float a = dt * diff * N * N;
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	// +1 po od 1 zaczynały się pętle, a nie od zera
	int j = (index / N) + 1;
	int i = (index % N) + 1;


	//kompiluje się czyli intellisense nie ogarnia
	cooperative_groups::grid_group g = cooperative_groups::this_grid();

	for (int k = 0; k < 20; k++) {
		g.sync();

		if (j > N) {
			return;
		}

		//for (int i = 1; i <= N; i++) {
		//	for (int j = 1; j <= N; j++) {
		x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] +
			x[IX(i, j - 1)] + x[IX(i, j + 1)])) / (1 + (4 * a));
		//	}
		//}
		set_bnd(N, b, x);
	}
}

__device__ void advect(int N, int b, float* d, float* d0, float* u, float* v, float dt)
{
	cooperative_groups::grid_group g = cooperative_groups::this_grid();

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// +1 po od 1 zaczynały się pętle, a nie od zera
	int j = (index / N) + 1;
	int i = (index % N) + 1;

	if (j > N) {
		return;
	}

	int i0, j0, i1, j1;
	float x, y, s0, t0, s1, t1, dt0;
	dt0 = dt * (float)N;
	//for (int i = 1; i <= N; i++) {
	//	for (int j = 1; j <= N; j++) {

			x = (float)i - dt0 * u[IX(i, j)];
			if (x < 0.5) x = 0.5f;
			if (x > N + 0.5) x = N + 0.5f;
			i0 = (int)x;
			i1 = i0 + 1;

			//proporcje ile gę┌stości wylądowało z których komórek
			s1 = x - (float)i0;
			s0 = 1.0f - s1;

			y = (float)j - dt0 * v[IX(i, j)];
			if (y < 0.5) y = 0.5f;
			if (y > N + 0.5) y = N + 0.5f;
			j0 = (int)y;
			j1 = j0 + 1;

			//proporcje ile gęstości wylądowało z których komórek
			t1 = y - (float)j0;
			t0 = 1.0f - t1;

			d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
				s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);

	//	}
	//}
	set_bnd(N, b, d);
}

__device__ void project(int N, float* u, float* v, float* p, float* div)
{

	cooperative_groups::grid_group g = cooperative_groups::this_grid();
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// +1 po od 1 zaczynały się pętle, a nie od zera
	int j = (index / N) + 1;
	int i = (index % N) + 1;

	if (j > N) {
		return;
	}


	int k;
	float h;
	h = 1.0f / N;
	//for (i = 1; i <= N; i++) {
	//	for (j = 1; j <= N; j++) {
			div[IX(i, j)] = -0.5f * h * (u[IX(i + 1, j)] - u[IX(i - 1, j)] +
				v[IX(i, j + 1)] - v[IX(i, j - 1)]);
			p[IX(i, j)] = 0.0f;
	//	}
	//}
	set_bnd(N, 0, div);
	set_bnd(N, 0, p);


	for (k = 0; k < 20; k++) {
		g.sync();
		//for (i = 1; i <= N; i++) {
		//	for (j = 1; j <= N; j++) {
				p[IX(i, j)] = (div[IX(i, j)] + p[IX(i - 1, j)] + p[IX(i + 1, j)] +
					p[IX(i, j - 1)] + p[IX(i, j + 1)]) / 4;
		//	}
		//}
		set_bnd(N, 0, p);
	}
	//for (i = 1; i <= N; i++) {
	//	for (j = 1; j <= N; j++) {
			u[IX(i, j)] -= 0.5f * (p[IX(i + 1, j)] - p[IX(i - 1, j)]) / h;
			v[IX(i, j)] -= 0.5f * (p[IX(i, j + 1)] - p[IX(i, j - 1)]) / h;
	//	}
	//}
	set_bnd(N, 1, u);
	set_bnd(N, 2, v);
}

__global__ void cuda_NextFrame(int N, float* d_dens, float* d_dens_prev, float* d_u, float* d_v, float* d_u_prev, float* d_v_prev, float visc, float diff , float dt) {
	//vel_step(size, d_u, d_v, d_u_prev, d_v_prev, visc, dt);
	
	diffuse(N, 1, d_u_prev, d_u, visc, dt);
	diffuse(N, 2, d_v_prev, d_v, visc, dt);
	project(N, d_u_prev, d_v_prev, d_u, d_v);
	advect(N, 1, d_u, d_u_prev, d_u_prev, d_v_prev, dt);
	advect(N, 2, d_v, d_v_prev, d_u_prev, d_v_prev, dt);
	project(N, d_u, d_v, d_u_prev, d_v_prev);
	
	
	//dens_step(size, d_dens, d_dens_prev, d_u, d_v, diff, dt);

	diffuse(N, 0, d_dens_prev, d_dens, diff, dt);
	advect(N, 0, d_dens, d_dens_prev, d_u, d_v, dt);
}

__global__ void addDensity(int N, float* d_dens, int x1, int x2, int y1, int y2, float dens)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int deltay = y2 - y1;
	int deltax = x2 - x1;
	int j = (index / deltax);
	int i = (index % deltax);

	if (j > deltay)
	{
		return;
	}

	d_dens[IX(x1 + i,y1 + j)] += dens;
	
	if (d_dens[IX(x1 + i, y1 + j)] > 1)
	{
		d_dens[IX(x1 + i, y1 + j)] = 1;
	}
}

__global__ void addVelocity(int N, float* d_u, float* d_v, int x, int y, int r, float u_velocity, float v_velocity)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int j = (index / (r*2));
	int i = (index % (r*2));
	int r_square = r * r;

	if (N > x + i && x + i > 0 && N > y + j && y + j > 0 && i * i + j * j < r_square)
	{
		d_u[IX(x + i, y + j)] += u_velocity;
		d_v[IX(x + i, y + j)] += v_velocity;
	}
}
//__device__ void vel_step(int N, float* u, float* v, float* u0, float* v0, float visc, float dt)
//{
//	//add_source(N, u, u0, dt); add_source(N, v, v0, dt);
//	diffuse(N, 1, u0, u, visc, dt);
//	diffuse(N, 2, v0, v, visc, dt);
//	project(N, u0, v0, u, v);
//	advect(N, 1, u, u0, u0, v0, dt);
//	advect(N, 2, v, v0, u0, v0, dt);
//	project(N, u, v, u0, v0);
//}

//__device__ void dens_step(int N, float* x, float* x0, float* u, float* v, float diff, float dt)
//{
//	//add_source(N, x, x0, dt);
//	//print(x, "poczatek:");
//	diffuse(N, 0, x0, x, diff, dt);
//	//print(x, "po dyfuzji:");
//	//adwekcja zmienia ilość2 cieczy
//	advect(N, 0, x, x0, u, v, dt);
//	//print(x, "po advekcji:");
//}